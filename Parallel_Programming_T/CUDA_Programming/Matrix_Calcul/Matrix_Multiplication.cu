#include "hip/hip_runtime.h"
#include <cuda_tid.h>

#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#define ROW_SIZE 32
#define COL_SIZE 32
#define K_SIZE 128

#define WORK_LOAD 1024
#define MAT_SIZE_A (ROW_SIZE * K_SIZE)
#define MAT_SIZE_B (K_SIZE * COL_SIZE)
#define MAT_SIZE_C (ROW_SIZE * COL_SIZE)

#define memset_Zero(_p, _type, _size) memset(_p, 0, sizeof(_type) * _size)
#define dMem_Alloc(_p, _type, _size) hipMalloc(&_p, sizeof(_type) * _size)

void gen_Input_Matrices(float (*_a_mat)[K_SIZE], float (*_b_mat)[COL_SIZE]);
bool check_result(float (*_host_c_mat)[COL_SIZE], float (*_device_c_mat)[COL_SIZE]);

#if (K_SIZE <= 128)
__global__ void matMul_kernel(float* _a_mat, float* _b_mat, float* _c_mat)
{
    int row = threadIdx.y;
    int col = threadIdx.x;
    int index = row * blockDim.x + col;

    _c_mat[index] = 0;
    
    for (int k = 0; k < K_SIZE; k++)
        for (int i = 0; i < WORK_LOAD; i++)
            _c_mat[index] += _a_mat[row * K_SIZE + k] * _b_mat[col + k * COL_SIZE];
}

__global__ void matMul_kernel_rowx(float* _a_mat, float* _b_mat, float* _c_mat)
{
    int row = threadIdx.x;
    int col = threadIdx.y;
    int index = row * blockDim.y + col;

    _c_mat[index] = 0;
    
    for (int k = 0; k < K_SIZE; k++)
        for (int i = 0; i < WORK_LOAD; i++)
            _c_mat[index] += _a_mat[row * K_SIZE + k] * _b_mat[col + k * COL_SIZE];
}

__global__ void matMul_kernel_shared(float* _a_mat, float* _b_mat, float* _c_mat)
{
    int row = threadIdx.y;
    int col = threadIdx.x;
    int index = row * blockDim.x + col;

    __shared__ float s_a[ROW_SIZE][K_SIZE];
    __shared__ float s_b[K_SIZE][COL_SIZE];

    for (int k = 0; k < K_SIZE; k++)
    {
        s_a[row][k] = _a_mat[row * K_SIZE + k];
        s_b[k][col] = _b_mat[col + k * COL_SIZE];
    }

    __syncthreads();

    _c_mat[index] = 0;
    for (int k = 0; k < K_SIZE; k++)
        for (int i = 0; i < WORK_LOAD; i++)
            _c_mat[index] += s_a[row][k] * s_b[k][col];
    
}
#endif

__global__ void matMul_kernel_register(float* _a_mat, float* _b_mat, float* _c_mat)
{
    int row = threadIdx.y;
    int col = threadIdx.x;
    int index = row * blockDim.x + col;

    float s_c = 0;
    
    for (int k = 0; k < K_SIZE; k++)
        for (int i = 0; i < WORK_LOAD; i++)
            s_c += _a_mat[row * K_SIZE + k] * _b_mat[col + k * COL_SIZE];

    _c_mat[index] = s_c;
}

__global__ void matMul_kernel_shared_c(float* _a_mat, float* _b_mat, float* _c_mat)
{
    int row = threadIdx.y;
    int col = threadIdx.x;
    int index = row * blockDim.x + col;

    __shared__ float s_c[MAT_SIZE_C];
    
    s_c[index] = 0;
    for (int k = 0; k < K_SIZE; k++)
        for (int i = 0; i < WORK_LOAD; i++)
            s_c[index] += _a_mat[row * K_SIZE + k] * _b_mat[col + k * COL_SIZE];
            
    _c_mat[index] = s_c[index];
}

int main(void)
{
    float a_mat[ROW_SIZE][K_SIZE] = { .0, };
    float b_mat[K_SIZE][COL_SIZE] = { .0, };
    float host_c_mat[ROW_SIZE][COL_SIZE] = { .0, };
    float device_c_mat[ROW_SIZE][COL_SIZE] = { .0, };

    float* da_mat = nullptr;
    float* db_mat = nullptr;
    float* dc_mat = nullptr;

    clock_t start_time = NULL;
    clock_t end_time = NULL;

    memset_Zero(a_mat, float, MAT_SIZE_A);
    memset_Zero(b_mat, float, MAT_SIZE_B);
    memset_Zero(host_c_mat, float, MAT_SIZE_C);
    memset_Zero(device_c_mat, float, MAT_SIZE_C);

    dMem_Alloc(da_mat, float, MAT_SIZE_A);
    dMem_Alloc(db_mat, float, MAT_SIZE_B);
    dMem_Alloc(dc_mat, float, MAT_SIZE_C);

    gen_Input_Matrices(a_mat, b_mat);

    start_time = clock();
    for(int r = 0; r < ROW_SIZE; r++)
        for(int c = 0; c < COL_SIZE; c++)
            for(int k = 0; k < K_SIZE; k++)
                for(int i = 0; i < WORK_LOAD; i++)
                host_c_mat[r][c] += a_mat[r][k] * b_mat[k][c];
    end_time = clock();
    printf("HOST_MULTIPLICATION >> %lf s\n", (double)(end_time-start_time) / CLOCKS_PER_SEC);

    start_time = clock();
    hipMemcpy(da_mat, a_mat, sizeof(float) * MAT_SIZE_A, hipMemcpyHostToDevice);
    hipMemcpy(db_mat, b_mat, sizeof(float) * MAT_SIZE_B, hipMemcpyHostToDevice);
    end_time = clock();
    printf("DATA TRANSFER (HOST -> DEVICE) >> %lf s\n", (double)(end_time-start_time) / CLOCKS_PER_SEC);

    dim3 block_dim(COL_SIZE, ROW_SIZE);
    dim3 block_dim_row_x(ROW_SIZE, COL_SIZE);
    dim3 grid_dim(1);
#if (K_SIZE <= 128)
    start_time = clock();
    matMul_kernel<<<grid_dim, block_dim>>>(da_mat, db_mat, dc_mat);
    hipDeviceSynchronize();
    end_time = clock();
    printf("KERNEL_MULTIPLICATION (GLOBAL) >> %lf s\n", (double)(end_time-start_time) / CLOCKS_PER_SEC);

    start_time = clock();
    matMul_kernel_rowx<<<grid_dim, block_dim_row_x>>>(da_mat, db_mat, dc_mat);
    hipDeviceSynchronize();
    end_time = clock();
    printf("KERNEL_MULTIPLICATION (GLOBAL_ROW_X) >> %lf s\n", (double)(end_time-start_time) / CLOCKS_PER_SEC);
    
    start_time = clock();
    matMul_kernel_shared<<<grid_dim, block_dim>>>(da_mat, db_mat, dc_mat);
    hipDeviceSynchronize();
    end_time = clock();
    printf("KERNEL_MULTIPLICATION (SHARED) >> %lf s\n", (double)(end_time-start_time) / CLOCKS_PER_SEC);
#endif

    start_time = clock();
    matMul_kernel_register<<<grid_dim, block_dim>>>(da_mat, db_mat, dc_mat);
    hipDeviceSynchronize();
    end_time = clock();
    printf("KERNEL_MULTIPLICATION (REGISTER) >> %lf s\n", (double)(end_time-start_time) / CLOCKS_PER_SEC);


    start_time = clock();
    matMul_kernel_shared_c<<<grid_dim, block_dim>>>(da_mat, db_mat, dc_mat);
    hipDeviceSynchronize();
    end_time = clock();
    printf("KERNEL_MULTIPLICATION (SHARED_C) >> %lf s\n", (double)(end_time-start_time) / CLOCKS_PER_SEC);

    start_time = clock();
    hipMemcpy(device_c_mat, dc_mat, sizeof(float) * MAT_SIZE_C, hipMemcpyDeviceToHost);
    end_time = clock();
    printf("DATA TRANSFER (DEVICE -> HOST) >> %lf s\n", (double)(end_time-start_time) / CLOCKS_PER_SEC);

    if(!check_result(host_c_mat, device_c_mat)) printf("!!!!!!Result is not correct!!!!!!\n");
    else printf("!Result is correct!\n");

    return 0;
}

void gen_Input_Matrices(float (*_a_mat)[K_SIZE], float (*_b_mat)[COL_SIZE])
{
    for (int r = 0; r < ROW_SIZE; r++)
        for (int k = 0; k < K_SIZE; k++)
            _a_mat[r][k] = rand() % 100;

    for (int k = 0; k < K_SIZE; k++)
        for (int c = 0; c < COL_SIZE; c++)
            _b_mat[k][c] = rand() % 100;
}

bool check_result(float (*_host_c_mat)[COL_SIZE], float (*_device_c_mat)[COL_SIZE])
{
    bool is_correct = true;

    float* p_host_c = &_host_c_mat[0][0];
    float* p_device_c = &_device_c_mat[0][0];

    for (int i = 0; i < MAT_SIZE_C; i++)
    {
        if (p_host_c[i] != p_device_c[i])
        {
            printf("[%d] %.2f, %.2f\n", i, p_host_c[i], p_device_c[i]);
            is_correct = false;
            break;
        }
    }
    
    return is_correct;
}