#include "hip/hip_runtime.h"
#include <cuda_tid.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>


#define NUM_BLOCK (128 * 1024)
#define NUM_T_IN_B 1024
#define ARRAY_SIZE (NUM_T_IN_B * NUM_BLOCK)

#define NUM_STREAMS 2

__global__ void my_kernel(int* _input, int* _output)
{
    int index = GRID_1D_BLOCK_1D_TID;

    int temp = 0;
    for (int i = 0; i < 250; i++)
        temp = (temp + _input[index] * 5) % 10;

    _output[index] = temp;
}

int main(void)
{
    int* in = nullptr;
    int* out = nullptr;

    int* din = nullptr;
    int* dout = nullptr;

    hipHostMalloc(&in, sizeof(int)*ARRAY_SIZE);
    memset(in, 0, sizeof(int)*ARRAY_SIZE);
    hipHostMalloc(&out, sizeof(int)*ARRAY_SIZE);
    memset(out, 0, sizeof(int)*ARRAY_SIZE);

    hipMalloc(&din, sizeof(int)*ARRAY_SIZE);
    hipMalloc(&dout, sizeof(int)*ARRAY_SIZE);

    for (int i = 0; i < ARRAY_SIZE; i++)
        in[i] = rand() % 10;

    hipStream_t stream[NUM_STREAMS];
    hipEvent_t start[NUM_STREAMS];
    hipEvent_t end[NUM_STREAMS];

    for (int i = 0; i < NUM_STREAMS; i++)
    {
        hipStreamCreate(&stream[i]);
        hipEventCreate(&start[i]);
        hipEventCreate(&end[i]);
    }

    int chunck_size = ARRAY_SIZE / NUM_STREAMS;
    
    for (int i = 0; i < NUM_STREAMS; i++)
    {
        int offset = chunck_size * i;
        hipEventRecord(start[i], stream[i]);
        hipMemcpyAsync(din + offset, in + offset, sizeof(int)*chunck_size, hipMemcpyHostToDevice, stream[i]);
        my_kernel<<<NUM_BLOCK / NUM_STREAMS, NUM_T_IN_B, 0, stream[i]>>>(din + offset, dout + offset);
        hipMemcpyAsync(out + offset, dout + offset, sizeof(int)*chunck_size, hipMemcpyDeviceToHost, stream[i]);
        hipEventRecord(end[i], stream[i]);
    }    
    
    hipDeviceSynchronize();

    for (int i = 0; i < NUM_STREAMS; i++)
    {
        float time = .0f;
        hipEventElapsedTime(&time, start[i], end[i]);
        printf("Stream[%d] Time >> %f ms\n", i, time);
    }

    for (int i = 0; i < NUM_STREAMS; i++)
    {
        hipStreamDestroy(stream[i]);
        hipEventDestroy(start[i]);
        hipEventDestroy(end[i]);
    }
    
    hipFree(din);
    hipFree(dout);
    
    hipHostFree(in);
    hipHostFree(out);

    return 0;
}