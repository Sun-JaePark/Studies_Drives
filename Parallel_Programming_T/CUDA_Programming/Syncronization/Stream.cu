#include "hip/hip_runtime.h"
#include <cuda_tid.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>


#define NUM_BLOCK (128 * 1024)
#define NUM_T_IN_B 1024
#define ARRAY_SIZE (NUM_T_IN_B * NUM_BLOCK)
#define WORK_LOAD 10240

#define NUM_STREAMS 1024

__global__ void my_kernel(int* _input, int* _output)
{
    int index = GRID_1D_BLOCK_1D_TID;

    int temp = 0;
    for (int i = 0; i < WORK_LOAD; i++)
        temp = (temp + _input[index] * 5) % 10;

    _output[index] = temp;
}

int main(void)
{
    clock_t start = NULL;
    clock_t end = NULL;

    int* in = nullptr;
    int* out = nullptr;

    int* din = nullptr;
    int* dout = nullptr;

    hipHostMalloc(&in, sizeof(int)*ARRAY_SIZE);
    memset(in, 0, sizeof(int)*ARRAY_SIZE);
    hipHostMalloc(&out, sizeof(int)*ARRAY_SIZE);
    memset(out, 0, sizeof(int)*ARRAY_SIZE);

    hipMalloc(&din, sizeof(int)*ARRAY_SIZE);
    hipMalloc(&dout, sizeof(int)*ARRAY_SIZE);

    for (int i = 0; i < ARRAY_SIZE; i++)
        in[i] = rand() % 10;

    start = clock();
    hipMemcpy(din, in, sizeof(int)*ARRAY_SIZE, hipMemcpyHostToDevice);
    my_kernel<<<NUM_BLOCK, NUM_T_IN_B>>>(din, dout);
    hipMemcpy(out, dout, sizeof(int)*ARRAY_SIZE, hipMemcpyDeviceToHost);
    end = clock();
    
    printf("NULL_STREAM time >> %lf s\n\n", (double)(end-start) / CLOCKS_PER_SEC);

    hipStream_t stream[NUM_STREAMS];

    for (int i = 0; i < NUM_STREAMS; i++)
        hipStreamCreate(&stream[i]);

    int chunck_size = ARRAY_SIZE / NUM_STREAMS;
    
    start = clock();
    for (int i = 0; i < NUM_STREAMS; i++)
    {
        int offset = chunck_size * i;
        hipMemcpyAsync(din + offset, in + offset, sizeof(int)*chunck_size, hipMemcpyHostToDevice, stream[i]);
        my_kernel<<<NUM_BLOCK / NUM_STREAMS, NUM_T_IN_B, 0, stream[i]>>>(din + offset, dout + offset);
        hipMemcpyAsync(out + offset, dout + offset, sizeof(int)*chunck_size, hipMemcpyDeviceToHost, stream[i]);
    }    
    
    hipDeviceSynchronize();
    end = clock();

    printf("NON_NULL_STREAM time >> %lf s\n\n", (double)(end-start) / CLOCKS_PER_SEC);

    for (int i = 0; i < NUM_STREAMS; i++)
        hipStreamDestroy(stream[i]);
    
    hipFree(din);
    hipFree(dout);
    
    hipHostFree(in);
    hipHostFree(out);

    return 0;
}