#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

int main(void)
{
    int ngpus = 0;
    hipGetDeviceCount(&ngpus);

    for (int i = 0; i < ngpus; i++)
    {
        hipDeviceProp_t devProp;

        hipGetDeviceProperties(&devProp, i);
        printf("Device[%d] (%s) compute capability : %d.%d.\n", i, devProp.name, devProp.major, devProp.minor);
        printf("maxThreadsDim[1] // [2] // [3] : %d, %d, %d\n", devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
        printf("maxGridSize : %d\n", *devProp.maxGridSize);
        printf("maxThreadsPerBlock : %d\n", devProp.maxThreadsPerBlock);
    }
    
    return 0;
}