#include "hip/hip_runtime.h"
#include <cuda_tid.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void threadCounting_Ver1(int* _d)
{
    (*_d)++;
}

__global__ void threadCounting_Ver2(int* _d)
{
    atomicAdd(_d, 1);
}

__global__ void threadCounting_Ver3(int* _d)
{
    __shared__ int sa;

    if(threadIdx.x == 0) sa = 0;

    __syncthreads();

    atomicAdd(&sa, 1);

    __syncthreads();

    if(threadIdx.x == 0) atomicAdd(_d, sa);
}

int main(int argc, char** argv)
{
    clock_t start = NULL;
    clock_t end = NULL;

    int a = 0;
    int *d1 = nullptr;
    int *d2 = nullptr;
    int *d3 = nullptr;

    hipMalloc((void**)&d1, sizeof(int));
    hipMalloc((void**)&d2, sizeof(int));
    hipMalloc((void**)&d3, sizeof(int));
    hipMemset(d1, 0, sizeof(int) * 1);
    hipMemset(d2, 0, sizeof(int) * 1);
    hipMemset(d3, 0, sizeof(int) * 1);

    dim3 block_size(512);
    dim3 grid_size(10240);

    start = clock();
    threadCounting_Ver1<<<grid_size, block_size>>>(d1);
    hipDeviceSynchronize();
    end = clock();

    hipMemcpy(&a, d1, sizeof(int), hipMemcpyDeviceToHost);

    printf("No_Sync res >> %d\n", a);
    printf("No_Sync time >> %lf s\n\n", (double)(end-start) / CLOCKS_PER_SEC);

    start = clock();
    threadCounting_Ver2<<<grid_size, block_size>>>(d2);
    hipDeviceSynchronize();
    end = clock();

    hipMemcpy(&a, d2, sizeof(int), hipMemcpyDeviceToHost);

    printf("Global_Atomic res >> %d\n", a);
    printf("Global_Atomic time >> %lf s\n\n", (double)(end-start) / CLOCKS_PER_SEC);

    start = clock();
    threadCounting_Ver3<<<grid_size, block_size>>>(d3);
    hipDeviceSynchronize();
    end = clock();

    hipMemcpy(&a, d3, sizeof(int), hipMemcpyDeviceToHost);

    printf("Shared_Atomic res >> %d\n", a);
    printf("Shared_Atomic time >> %lf s\n", (double)(end-start) / CLOCKS_PER_SEC);

    return 0;
}
