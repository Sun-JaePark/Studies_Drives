#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctime>
#include <cuda_tid.h>

#define NUM_DATA 10240
#define BLOCK_MAX 1024
#define IS_HOST_CAL 0

__global__ void vec_Add(int* _a, int* _b, int* _c)
{
    int tID = GRID_1D_BLOCK_1D_TID;
    _c[tID] = _a[tID] + _b[tID];
}

int main(void)
{
    int* a = nullptr;
    int* b = nullptr;
    int* c = nullptr;

    int* d_a = nullptr;
    int* d_b = nullptr;
    int* d_c = nullptr;

    int mem_size = sizeof(int) * NUM_DATA;
    printf("%d elements, mem_size = %d bytes\n", NUM_DATA, mem_size);

    a = new int[NUM_DATA];
    b = new int[NUM_DATA];
    c = new int[NUM_DATA];

    memset(a, 0, mem_size);
    memset(b, 0, mem_size);
    memset(c, 0, mem_size);

    for (int i = 0; i < NUM_DATA; i++)
    {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

#if IS_HOST_CAL
    time_t start = time(NULL);

    for (int i = 0; i < NUM_DATA; i++)
        c[i] = a[i] + b[i];

    time_t end = time(NULL);
#else
    hipMalloc(&d_a, mem_size);
    hipMalloc(&d_b, mem_size);
    hipMalloc(&d_c, mem_size);

    hipMemcpy(d_a, a, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, mem_size, hipMemcpyHostToDevice);

    time_t start = time(NULL);
    dim3 block(BLOCK_MAX);
    dim3 grid((BLOCK_MAX + NUM_DATA - 1) / BLOCK_MAX);
    vec_Add<<<grid, block>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();
    time_t end = time(NULL);

    hipMemcpy(c, d_c, mem_size, hipMemcpyDeviceToHost);

    bool is_result = true;
    for (int i = 0; i < NUM_DATA; i++)
    {
        if((a[i] + b[i]) != c[i])
        {
            printf("[%d] The results is not matched! (%d, %d)\n", i, a[i] + b[i], c[i]);
            is_result = false;
        }
    }

    if(is_result) printf("GPU works well!\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
#endif
    delete[] a;
    delete[] b;
    delete[] c;

    printf("Total Time : [%lf] ms\n", (double)end-start);
    return 0;
}