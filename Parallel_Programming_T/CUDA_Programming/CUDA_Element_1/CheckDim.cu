#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctime>

__global__ void check_Index(void)
{
    printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n",
    threadIdx.x, threadIdx.y, threadIdx.z,
    blockIdx.x, blockIdx.y, blockIdx.z,
    blockDim.x, blockDim.y, blockDim.z,
    gridDim.x, gridDim.y, gridDim.z);
}

int main(int argc, char** argv)
{
    int n_elem = 40;
    dim3 block(5, 1, 2);
    int block_size = block.x * block.y * block.z;
    dim3 grid((n_elem + block_size - 1) / block_size);

    printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);

    check_Index <<<grid, block>>>();
    hipDeviceReset();

    return 0;
}